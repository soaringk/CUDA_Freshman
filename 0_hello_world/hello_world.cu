
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function that runs on the GPU
__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    // Launch the kernel on the GPU
    // <<<1, 1>>> specifies one block and one thread
    cuda_hello<<<1, 10>>>(); 

    // Synchronize the CPU and GPU to ensure the print statement finishes
    hipDeviceSynchronize(); 

    printf("Hello World from CPU!\n"); // This will print first
    return 0;
}
